#include "helper.h"
#include "SSFRendererImpl.h"
#include <GLFW\glfw3.h>
#include <glad\glad.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <cuda_gl_interop.h>

static float quadVertices[] = { // vertex attributes for a quad that fills the entire screen in Normalized Device Coordinates.
    // positions   // texCoords
    -1.0f,  1.0f,  0.0f, 1.0f,
    -1.0f, -1.0f,  0.0f, 0.0f,
    1.0f, -1.0f,  1.0f, 0.0f,

    -1.0f,  1.0f,  0.0f, 1.0f,
    1.0f, -1.0f,  1.0f, 0.0f,
    1.0f,  1.0f,  1.0f, 1.0f
};

SSFRendererImpl::SSFRendererImpl(Camera *camera, int width, int height)
{
	m_niter = 4;

	/* TODO: consider how to handle resolution change */
	this->m_camera = camera;
	this->m_width = width;
	this->m_height = height;
	this->m_pi = camera->getProjectionInfo();

	/* Allocate depth / normal_D / H texture */
	glGenTextures(1, &d_depth);
	glGenTextures(1, &d_depth_r);
	glGenTextures(1, &d_normal_D);
	glGenTextures(1, &d_H);

	glBindTexture(GL_TEXTURE_2D, d_normal_D);
	glTexImage2D(GL_TEXTURE_2D, 0, GL_RGBA32F, width, height, 0, GL_RGBA, GL_FLOAT, NULL);
	/* TODO: check effect of GL_NEAREST */
	glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_MIN_FILTER, GL_NEAREST);
	glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_MAG_FILTER, GL_NEAREST);
	checkGLErr();
	glBindTexture(GL_TEXTURE_2D, d_depth);
	glTexImage2D(GL_TEXTURE_2D, 0, GL_DEPTH_COMPONENT32F, width, height, 0, GL_DEPTH_COMPONENT, GL_FLOAT, NULL);
	glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_MIN_FILTER, GL_NEAREST);
	glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_MAG_FILTER, GL_NEAREST);
	checkGLErr();
	glBindTexture(GL_TEXTURE_2D, d_depth_r);
	glTexImage2D(GL_TEXTURE_2D, 0, GL_R32F, width, height, 0, GL_RED, GL_FLOAT, NULL);
	glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_MIN_FILTER, GL_NEAREST);
	glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_MAG_FILTER, GL_NEAREST);
	checkGLErr();
	glBindTexture(GL_TEXTURE_2D, d_H);
	glTexImage2D(GL_TEXTURE_2D, 0, GL_R32F, width, height, 0, GL_RED, GL_FLOAT, NULL);
	glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_MIN_FILTER, GL_NEAREST);
	glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_MAG_FILTER, GL_NEAREST);
	checkGLErr();

	/* TODO: Bind texture to CUDA resource */
	//checkCudaErrors(hipGraphicsGLRegisterImage(&dcr_normal_D, d_normal_D, GL_TEXTURE_2D, hipGraphicsRegisterFlagsNone));
	/* CUDA does not support interop with GL_DEPTH_COMPONENT texture ! 
	 * As a workaround, first render to a depth texture (d_depth)
	 * then copy depth texture to a color texture (d_depth_r), which contains only red channel 
	 */
	//checkCudaErrors(hipGraphicsGLRegisterImage(&dcr_depth, d_depth_r, GL_TEXTURE_2D, hipGraphicsRegisterFlagsNone));
	//checkCudaErrors(hipGraphicsGLRegisterImage(&dcr_H, d_H, GL_TEXTURE_2D, hipGraphicsRegisterFlagsNone));

	/* Allocate framebuffer & Binding depth texture */
	glGenFramebuffers(1, &d_fbo);
	glBindFramebuffer(GL_FRAMEBUFFER, d_fbo);
	glBindTexture(GL_TEXTURE_2D, d_depth);
	glFramebufferTexture2D(GL_FRAMEBUFFER, GL_DEPTH_ATTACHMENT, GL_TEXTURE_2D, d_depth, 0);

	/* Attach one color buffer, this is mandatory */
	uint colorTex;
	glGenTextures(1, &colorTex);
	glBindTexture(GL_TEXTURE_2D, colorTex);
	glTexImage2D(GL_TEXTURE_2D, 0, GL_RGB, width, height, 0, GL_RGB, GL_UNSIGNED_BYTE, NULL);
	glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_MIN_FILTER, GL_LINEAR);
	glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_MAG_FILTER, GL_LINEAR);
	glFramebufferTexture2D(GL_FRAMEBUFFER, GL_COLOR_ATTACHMENT0, GL_TEXTURE_2D, colorTex, 0);
	
	glFramebufferTexture2D(GL_FRAMEBUFFER, GL_COLOR_ATTACHMENT1, GL_TEXTURE_2D, d_depth_r, 0);
	glFramebufferTexture2D(GL_FRAMEBUFFER, GL_COLOR_ATTACHMENT2, GL_TEXTURE_2D, d_normal_D, 0);
	glFramebufferTexture2D(GL_FRAMEBUFFER, GL_COLOR_ATTACHMENT3, GL_TEXTURE_2D, d_H, 0);

	checkFramebufferComplete();
	checkGLErr();

	glBindFramebuffer(GL_FRAMEBUFFER, 0);

	/* Load shaders */
	m_s_get_depth = new Shader(Filename("SSFget_depth.v.glsl"), Filename("SSFget_depth.f.glsl"));
	m_s_put_depth = new Shader(Filename("SSFput_depth.v.glsl"), Filename("SSFput_depth.f.glsl"));
	m_s_restore_normal = new Shader(Filename("SSFrestore_normal.v.glsl"), Filename("SSFrestore_normal.f.glsl"));
	m_s_computeH = new Shader(Filename("SSFcomputeH.v.glsl"), Filename("SSFcomputeH.f.glsl"));

	/* Load quad vao */
	uint quad_vbo;
	glGenVertexArrays(1, &m_quad_vao);
	glGenBuffers(1, &quad_vbo);
	glBindVertexArray(m_quad_vao);
	glBindBuffer(GL_ARRAY_BUFFER, quad_vbo);
	glBufferData(GL_ARRAY_BUFFER, sizeof(quadVertices), quadVertices, GL_STATIC_DRAW);
	glEnableVertexAttribArray(0);
	glVertexAttribPointer(0, 2, GL_FLOAT, GL_FALSE, 4 * sizeof(float), (void*)0);
	glEnableVertexAttribArray(1);
	glVertexAttribPointer(1, 2, GL_FLOAT, GL_FALSE, 4 * sizeof(float), (void*)(2 * sizeof(float)));
}

void SSFRendererImpl::destroy() {
	// if (!dc_depth) return;
	/* TODO */
}

void SSFRendererImpl::renderDepth() {
	/* Render to framebuffer */
	glBindFramebuffer(GL_FRAMEBUFFER, d_fbo);

	m_s_get_depth->use();
	m_camera->use(Shader::now());

	m_s_get_depth->setUnif("pointRadius", 50.f);

	glEnable(GL_DEPTH_TEST);
	glBindVertexArray(p_vao);

	if (glCheckFramebufferStatus(GL_FRAMEBUFFER) != GL_FRAMEBUFFER_COMPLETE)
		fexit(-1, "Framebuffer not complete\n");

	glClear(GL_DEPTH_BUFFER_BIT);
	glDrawArrays(GL_POINTS, 0, m_nparticle);
	glBindFramebuffer(GL_FRAMEBUFFER, 0);

	/* Copy depth into a color texture */
	glCopyImageSubData(
		d_depth, GL_TEXTURE_2D, 0, 0, 0, 0,
		d_depth_r, GL_TEXTURE_2D, 0, 0, 0, 0,
		m_width, m_height, 1);
	checkGLErr();
}

void SSFRendererImpl::renderPlane() {

	/* Draw depth in greyscale */
	m_s_put_depth->use();
	m_camera->use(Shader::now());

	ProjectionInfo i = m_camera->getProjectionInfo();
	m_s_put_depth->setUnif("projZNear", i.n);
	m_s_put_depth->setUnif("projZFar", i.f);

	glDisable(GL_DEPTH_TEST);
	glBindVertexArray(m_quad_vao);
	glBindTexture(GL_TEXTURE_2D, d_depth_r);
	glDrawArrays(GL_TRIANGLES, 0, 6);
	glEnable(GL_DEPTH_TEST);
}

void SSFRendererImpl::render(uint p_vao, int nparticle) {

	this->p_vao = p_vao;
	this->m_nparticle = nparticle;

	renderDepth();

	// mapResources();
	for (int i = 0; i < m_niter; i++) {
		restoreNormal();
		computeH();
		updateDepth();
	}
	// unmapResources();

	renderPlane();
}

void SSFRendererImpl::mapResources() {
	checkCudaErrors(hipGraphicsMapResources(1, &dcr_depth, 0));
	checkCudaErrors(hipGraphicsMapResources(1, &dcr_normal_D, 0));
	checkCudaErrors(hipGraphicsMapResources(1, &dcr_H, 0));

	size_t size;
	checkCudaErrors(hipGraphicsResourceGetMappedPointer((void**)dc_depth, &size, dcr_depth));
	checkCudaErrors(hipGraphicsResourceGetMappedPointer((void**)dc_normal_D, &size, dcr_normal_D));
	checkCudaErrors(hipGraphicsResourceGetMappedPointer((void**)dc_H, &size, dcr_H));
}

void SSFRendererImpl::unmapResources() {
	checkCudaErrors(hipGraphicsUnmapResources(1, &dcr_depth, 0));
	checkCudaErrors(hipGraphicsUnmapResources(1, &dcr_normal_D, 0));
	checkCudaErrors(hipGraphicsUnmapResources(1, &dcr_H, 0));

	/* TODO: check if need unregister resource using hipGraphicsUnregisterResource() */
}

void SSFRendererImpl::restoreNormal() {
	/*int block_size = 256;
	int grid_size = ceilDiv(m_nparticle, block_size);*/

	glBindFramebuffer(GL_FRAMEBUFFER, d_fbo);
	glDisable(GL_BLEND);

	m_s_restore_normal->use();
	m_camera->use(Shader::now());

	ProjectionInfo i = m_camera->getProjectionInfo();
	m_s_restore_normal->setUnif("p_n", i.n);
	m_s_restore_normal->setUnif("p_f", i.f);
	m_s_restore_normal->setUnif("p_t", i.t);
	m_s_restore_normal->setUnif("p_r", i.r);
	m_s_restore_normal->setUnif("s_w", (float)m_width);
	m_s_restore_normal->setUnif("s_h", (float)m_height);

	glDisable(GL_DEPTH_TEST);

	GLenum bufs[] = { GL_COLOR_ATTACHMENT2 /* d_normal_D */ };
	glDrawBuffers(1, bufs);
	
	glBindVertexArray(m_quad_vao);
	glBindTexture(GL_TEXTURE_2D, d_depth_r);
	glDrawArrays(GL_TRIANGLES, 0, 6);
	glBindFramebuffer(GL_FRAMEBUFFER, 0);
	glEnable(GL_BLEND);
}

void SSFRendererImpl::computeH() {
	/*int block_size = 256;
	int grid_size = ceilDiv(m_nparticle, block_size);*/
	glBindFramebuffer(GL_FRAMEBUFFER, d_fbo);
	glDisable(GL_BLEND);

	m_s_computeH->use();
	m_camera->use(Shader::now());

	ProjectionInfo i = m_camera->getProjectionInfo();
	m_s_computeH->setUnif("p_n", i.n);
	m_s_computeH->setUnif("p_f", i.f);
	m_s_computeH->setUnif("p_t", i.t);
	m_s_computeH->setUnif("p_r", i.r);
	m_s_computeH->setUnif("s_w", (float)m_width);
	m_s_computeH->setUnif("s_h", (float)m_height);

	glDisable(GL_DEPTH_TEST);

	GLenum bufs[] = { GL_COLOR_ATTACHMENT3 /* d_H */ };
	glDrawBuffers(1, bufs);

	glBindVertexArray(m_quad_vao);

	glActiveTexture(GL_TEXTURE0);
	glBindTexture(GL_TEXTURE_2D, d_depth_r);
	glActiveTexture(GL_TEXTURE1);
	glBindTexture(GL_TEXTURE_2D, d_normal_D);
	
	glDrawArrays(GL_TRIANGLES, 0, 6);
	glBindFramebuffer(GL_FRAMEBUFFER, 0);
	glEnable(GL_BLEND);
}

void SSFRendererImpl::updateDepth() {
	int block_size = 256;
	int grid_size = ceilDiv(m_nparticle, block_size);
}