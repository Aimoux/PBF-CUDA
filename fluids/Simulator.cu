#include "hip/hip_runtime.h"
#include "Simulator.h"
#include <thrust/transform_reduce.h>
#include <thrust/transform.h>
#include <thrust/iterator/zip_iterator.h>
#include <thrust/execution_policy.h>
#include <thrust/sort.h>
#include <thrust/functional.h>
#include <thrust/device_ptr.h>

#include "Simulator_kernel.cuh"

/* Helper functor */

struct helper_duplicate {
	__host__ __device__
	helper_duplicate() {}

	template <typename T> __device__
	thrust::tuple<T, T> operator()(const T t) { return thrust::make_tuple(t, t); }
};

struct getExtrema {
	typedef thrust::pair<float3, float3> PFF;

	__host__ __device__
	getExtrema() {}

	template <typename T> __device__
	T operator()(T a, T b) { 
		float3 &amax = thrust::get<0>(a), &amin = thrust::get<1>(a),
			   &bmax = thrust::get<0>(b), &bmin = thrust::get<1>(b);
		return thrust::make_tuple(
			make_float3(fmaxf(amax.x, bmax.x), fmaxf(amax.y, bmax.y), fmaxf(amax.z, bmax.z)),
			make_float3(fminf(amin.x, bmin.x), fminf(amin.y, bmin.y), fminf(amin.z, bmin.z)));
	}
};

struct getGridxyz {
	float3 llim;
	float h;
	int3 gridDim;
	__host__ __device__
	getGridxyz(const float3 &llim, const int3 &gridDim, float h) : llim(llim), gridDim(gridDim), h(2.f * h) {}

	__device__
	int3 operator()(float3 pos) {
		float3 diff = pos - llim;
		int x = diff.x / h, y = diff.y / h, z = diff.z / h;
		return make_int3(x, y, z);
	}
};

struct xyzToId {
	int3 gridDim;
	__host__ __device__
	xyzToId(const int3 &gridDim) : gridDim(gridDim) {}

	template <typename T> __device__
	int operator()(T x, T y, T z) {
		/*if (x < 0 || x >= gridDim.x || y < 0 || y >= gridDim.y || z < 0 || z >= gridDim.z)
			return -1;*/
		/* TODO */
		/*x = x & 15;
		y = y & 15;
		z = z & 15;*/
		return x * gridDim.y * gridDim.z + y * gridDim.z + z;
	}
};

struct getGridId {

	float3 llim;
	float h;
	int3 gridDim;
	__host__ __device__
	getGridId(const float3 &llim, const int3 &gridDim, float h) : llim(llim), gridDim(gridDim), h(2.f * h) {}

	template <typename T> __device__
	int operator()(T pos) {
		float3 diff = pos - llim;
		int x = diff.x / h, y = diff.y / h, z = diff.z / h;
		x = min(max(x, 0), gridDim.x);
		y = min(max(y, 0), gridDim.y);
		z = min(max(z, 0), gridDim.z);
		return (int)(x * gridDim.y * gridDim.z + y * gridDim.z + z);
	}
};

struct getPoly6 {
	float coef, h2, h9;
	getPoly6(float h) { 
		h2 = h * h;  
		float h3 = h2 * h;
		h9 = h3 * h3;
		coef = 315.f / (64.f * M_PI *  h9);
	}
	__device__
	float operator()(float r2) {
		return coef * (h2 - r2) * (h2 - r2) * (h2 - r2);
	}
};

struct getSpikyGrad {
	float h, h6, coef;
	getSpikyGrad(float h) : h(h) {
		h6 = h * h;
		h6 = h6 * h6 * h6;
		coef = -45.f / (M_PI * h6);
	}

	__device__
	float3 operator()(float3 r) {
		float rlen = length(r);
		return coef * (h - rlen) * (h - rlen) * normalize(r);
	}
};

struct h_updatePosition {
	float3 ulim, llim;
	h_updatePosition(float3 ulim, float3 llim) : ulim(ulim), llim(llim) {}

	template <typename T>
	__device__
	float3 operator()(T t) {
		float3 pos = thrust::get<0>(t), dpos = thrust::get<1>(t);
		pos += dpos;
		/* for now, project particles out of bound onto bounding box surface */
		pos.x = fmaxf(fminf(pos.x, ulim.x), llim.x);
		pos.y = fmaxf(fminf(pos.y, ulim.y), llim.y);
		pos.z = fmaxf(fminf(pos.z, ulim.z), llim.z);

		return pos;
	}
};

struct h_updateVelocity {
	float inv_dt;
	h_updateVelocity(float dt) : inv_dt(1.f / dt) {}

	template <typename T>
	__device__
	float3 operator()(T t) {
		float3 pos = thrust::get<0>(t), npos = thrust::get<1>(t);
		// printf("(%f,%f,%f) -> (%f,%f,%f)\n", pos.x, pos.y, pos.z, npos.x, npos.y, npos.z);
		return (npos - pos) * inv_dt;
	}
};

/* Intermedia steps */

void Simulator::advect()
{
	int block_size = 256;
	int grid_size = ceilDiv(m_nparticle, block_size);

	advect_kernel<<<grid_size, block_size>>>(
		dc_pos, dc_npos, 
		dc_vel, dc_nvel, 
		m_nparticle, m_dt, make_float3(0, 0, -m_gravity),
		m_ulim, m_llim);
}

void Simulator::buildGridHash()
{
	int block_size = 256;
	int grid_size = ceilDiv(m_nparticle, block_size);
	int smem = sizeof(uint) * (block_size + 2);

	thrust::device_ptr<float3> d_pos(dc_pos), d_vel(dc_vel), d_npos(dc_npos), d_nvel(dc_nvel);
	thrust::device_ptr<uint> d_gridId(dc_gridId);

	float3 diff = m_ulim - m_llim;
	m_gridHashDim = make_int3((int)ceilf(.5f * diff.x / m_h), (int)ceilf(.5f * diff.y / m_h), (int)ceilf(.5f * diff.z / m_h));
	/* Compute gridId for each particle */
	thrust::transform(
		d_npos, d_npos + m_nparticle,
		d_gridId,
		getGridId(m_llim, m_gridHashDim, m_h));

	/* sort (gridId, pos, vel) by gridId */
	thrust::sort_by_key(
		d_gridId, d_gridId + m_nparticle,
		thrust::make_zip_iterator(thrust::make_tuple(d_pos, d_vel, d_npos, d_nvel)));

	/* Compute [gradStart, gradEnd) */
	computeGridRange<<<grid_size, block_size, smem>>>(dc_gridId, dc_gridStart, dc_gridEnd, m_nparticle);

	// hipDeviceSynchronize();
	// getLastCudaError("Kernel execution failed: computeGridRange");

	/*printf("Counter=%d\n", *counter); 
	exit(0);*/
}

void Simulator::correctDensity() 
{
	/* Input:  
	 */

	int block_size = 256;
	int grid_size = ceilDiv(m_nparticle, block_size);

	// printf("maxStart = %u, maxMin = %u\n", arr[0], arr[1]);

	/// hipDeviceSynchronize();
	/* dc_npos -> dc_npos */
	computeLambda<<<grid_size, block_size>>>(
		dc_lambda, /*dc_gradl2,*/
		dc_gridId, dc_gridStart, dc_gridEnd,
		m_gridHashDim,
		dc_npos, m_nparticle, m_pho0, m_lambda_eps,
		/* getPoly6(m_h), getSpikyGrad(m_h), */ m_h,
		getGridxyz(m_llim, m_gridHashDim, m_h), xyzToId(m_gridHashDim));

	// hipDeviceSynchronize();
	// getLastCudaError("Kernel execution failed: computeLambda");

	m_coef_corr = -m_k_corr / powf(h_poly6(m_delta_q, m_h), m_n_corr);

	computedpos<<<grid_size, block_size>>>(
		dc_lambda, 
		dc_gridId, dc_gridStart, dc_gridEnd,
		m_gridHashDim,
		dc_npos, dc_dpos, m_nparticle, m_pho0, m_h, m_coef_corr, m_n_corr,
		getGridxyz(m_llim, m_gridHashDim, m_h), xyzToId(m_gridHashDim),
		m_ulim, m_llim);

}

void Simulator::updateVelocity() {
	/* Warn: assume dc_pos updates to dc_npos after correctDensity() */
	thrust::device_ptr<float3> d_pos(dc_pos), d_npos(dc_npos), d_nvel(dc_nvel);
	thrust::transform(
		thrust::make_zip_iterator(thrust::make_tuple(d_pos, d_npos)),
		thrust::make_zip_iterator(thrust::make_tuple(d_pos + m_nparticle, d_npos + m_nparticle)),
		d_nvel, h_updateVelocity(m_dt));
}